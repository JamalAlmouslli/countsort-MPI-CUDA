#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 256

__device__ unsigned long long deviceCount[65536];

__global__ void GenInputKernel(const int gpuCut, unsigned short* deviceData)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < gpuCut)
    {
        deviceData[tid] = (gpuCut - tid) & 65535;
    }
}

__global__ void InitCountKernel()
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    deviceCount[tid] = 0;
}

__global__ void CountKernel(const int gpuCut, const unsigned short* deviceData)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < gpuCut)
    {
        atomicAdd(&deviceCount[deviceData[tid]], 1ULL);
    }
}

static void CudaTest(const char* msg)
{
    hipError_t e;
    hipDeviceSynchronize();
    
    if (hipSuccess != (e = hipGetLastError()))
    {
        fprintf(stderr, "%s: %d\n", msg, e);
        fprintf(stderr, "%s\n", hipGetErrorString(e));
        exit(-1);
    }
}

unsigned short* allocGPU(const int gpuCut)
{
    unsigned short* deviceData;
    
    if (hipSuccess != hipMalloc((void **)&deviceData, gpuCut * sizeof(unsigned short)))
    {
        fprintf(stderr, "could not allocate GPU array\n");
        exit(-1);
    }
    
    return deviceData;
}

void deallocGPU(unsigned short* deviceData)
{
    hipFree(deviceData);
}

void runGPU(const int gpuCut, const int cpuCut, unsigned short* deviceData, unsigned short* hostData)
{
    InitCountKernel<<<65536 / THREADS, THREADS>>>();
    CudaTest("InitCountKernel failed\n");
    
    GenInputKernel<<<(gpuCut + THREADS - 1) / THREADS, THREADS>>>(gpuCut, deviceData);
    CudaTest("GenInputKernel failed\n");
    
    CountKernel<<<(gpuCut + THREADS - 1) / THREADS, THREADS>>>(gpuCut, deviceData);
    CudaTest("CountKernel failed\n");
    
    hipMemcpy(hostData + cpuCut, deviceData, gpuCut, hipMemcpyDeviceToHost);
}

